#include "hip/hip_runtime.h"
#include "Graph.h"
#define SQUARE_SIZE 512
void Graph::Component::CUDAFillEccentricities()
{
	vector<Vertex*> *i_members = &(this->members);

	hipMalloc(&i_members, sizeof(Vertex));

	hipMemcpy(i_members, this->members, sizeof(Vertex), hipMemcpyHostToDevice);

	for (int i = 0; i < members.size(); i += SQUARE_SIZE)
	{
		for (int y = i; y < i + SQUARE_SIZE; y++)
		{
			Vertex* vertex = members.at(i);
			vertex->eccentricity = CUDAFindEccentricity <<<1, SQUARE_SIZE >> > (y);
		}



	}
}

__global__ int Graph::Component::CUDAFindEccentricity(int startId)
{
	vector<int> pathDistance;
	pathDistance.assign(members.size() + 1, -1);
	pathDistance.at(startId) = 0;
	queue<Vertex*> q;
	Vertex* start = members.at(startId);
	q.push(start);

	Vertex* current = start;


	while (!q.empty())
	{
		current = q.front();
		q.pop();

		int size = current->neighbours.size();

		for (auto& neighbour : current->neighbours)
		{
			const int neighbourIndex = neighbour->index;
			if (pathDistance.at(neighbourIndex) == -1)
			{
				pathDistance.at(neighbourIndex) = pathDistance.at(current->index) + 1;
				q.push(neighbour);
			}
		}

	}
	return pathDistance.at(current->index);
}



/*

vector<int> pathDistance;
	pathDistance.assign(members.size() + 1, -1);
	pathDistance.at(startId) = 0;
	queue<Vertex*> q;
	Vertex* start = members.at(startId);
	q.push(start);

	Vertex* current = start;


	while (!q.empty())
	{
		current = q.front();
		q.pop();

		int size = current->neighbours.size();

		for (auto& neighbour : current->neighbours)
		{
			const int neighbourIndex = neighbour->index;
			if(pathDistance.at(neighbourIndex) == -1)
			{
				pathDistance.at(neighbourIndex) = pathDistance.at(current->index) + 1;
				q.push(neighbour);
			}
		}

	}
	return pathDistance.at(current->index);*/

/*
	for (int i = 0; i < members.size(); i++)
	{
		Vertex* vertex = members.at(i);
		if (vertex->eccentricity == -1)
		{

			vertex->eccentricity = findEccentricity(i);


		}
	}
*/